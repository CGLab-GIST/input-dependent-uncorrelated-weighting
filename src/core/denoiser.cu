#include "hip/hip_runtime.h"
/*
	Copyright (c) 2023 CGLab, GIST. All rights reserved.

	Redistribution and use in source and binary forms, with or without modification, 
	are permitted provided that the following conditions are met:

	- Redistributions of source code must retain the above copyright notice, 
	  this list of conditions and the following disclaimer.
	- Redistributions in binary form must reproduce the above copyright notice, 
	  this list of conditions and the following disclaimer in the documentation 
	  and/or other materials provided with the distribution.
	- Neither the name of the copyright holder nor the names of its contributors 
	  may be used to endorse or promote products derived from this software 
	  without specific prior written permission.

	THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" 
	AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE 
	IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE 
	ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE 
	LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL 
	DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR 
	SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
	CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, 
	OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE 
	OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>

#include <thrust/device_vector.h>
#include <thrust/reduce.h>

#include "denoiser.h"

#define IMAD(a, b, c)			( __mul24((a), (b)) + (c) )

inline int iDivUp(int a, int b) {
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

__forceinline__ __host__ __device__ void operator+=(float4 &a, float4 b) {
	a.x += b.x;
	a.y += b.y;
	a.z += b.z;
	a.w += b.w;
}

__forceinline__ __host__ __device__ float4 operator*(float b, float4 a) {
	return make_float4(b * a.x, b * a.y, b * a.z, b * a.w);
}

__forceinline__ __host__ __device__ float4 operator*(float4 a, float4 b) {
	return make_float4(a.x * b.x, a.y * b.y, a.z * b.z, a.w * b.w);
}

__forceinline__ __host__ __device__ float4 operator+(float4 a, float4 b) {
	return make_float4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}

__forceinline__ __host__ __device__ float4 operator-(float4 a, float4 b) {
	return make_float4(a.x - b.x, a.y - b.y, a.z - b.z, a.w - b.w);
}


__global__ void KernelCalcDenoisedVariance(float *outVar, 
	float *subImgsY, float *subImgsZ, int xSize, int ySize, int winSize, int spp, float paramGamma) {
	const int cx = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
	const int cy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);
	if (cx >= xSize || cy >= ySize)
		return;

	const int cIdx = cy * xSize + cx;
	const int halfWinSize = winSize / 2;
	const int winSizeSqr = winSize * winSize;
	const int colorDim = 3;
	const int nPix = colorDim * xSize * ySize;

	const int bufferIdxA0 = 0;
	const int bufferIdxA1 = 2;
	const int bufferIdxB0 = 1;
	const int bufferIdxB1 = 3;

	int cIdxA0 = bufferIdxA0 * nPix + cIdx * colorDim;
	int cIdxA1 = bufferIdxA1 * nPix + cIdx * colorDim;
	int cIdxB0 = bufferIdxB0 * nPix + cIdx * colorDim;
	int cIdxB1 = bufferIdxB1 * nPix + cIdx * colorDim;

	const float4& cImgYA0 = make_float4(subImgsY[cIdxA0 + 0], subImgsY[cIdxA0 + 1], subImgsY[cIdxA0 + 2], 0.f);
	const float4& cImgYA1 = make_float4(subImgsY[cIdxA1 + 0], subImgsY[cIdxA1 + 1], subImgsY[cIdxA1 + 2], 0.f);
	const float4& cImgYB0 = make_float4(subImgsY[cIdxB0 + 0], subImgsY[cIdxB0 + 1], subImgsY[cIdxB0 + 2], 0.f);
	const float4& cImgYB1 = make_float4(subImgsY[cIdxB1 + 0], subImgsY[cIdxB1 + 1], subImgsY[cIdxB1 + 2], 0.f);
	const float4& cImgZA0 = make_float4(subImgsZ[cIdxA0 + 0], subImgsZ[cIdxA0 + 1], subImgsZ[cIdxA0 + 2], 0.f);
	const float4& cImgZA1 = make_float4(subImgsZ[cIdxA1 + 0], subImgsZ[cIdxA1 + 1], subImgsZ[cIdxA1 + 2], 0.f);
	const float4& cImgZB0 = make_float4(subImgsZ[cIdxB0 + 0], subImgsZ[cIdxB0 + 1], subImgsZ[cIdxB0 + 2], 0.f);
	const float4& cImgZB1 = make_float4(subImgsZ[cIdxB1 + 0], subImgsZ[cIdxB1 + 1], subImgsZ[cIdxB1 + 2], 0.f);
	
	const float4 cImgYA = 0.5f * (cImgYA0 + cImgYA1);
	const float4 cImgYB = 0.5f * (cImgYB0 + cImgYB1);
	const float4 cImgZA = 0.5f * (cImgZA0 + cImgZA1);
	const float4 cImgZB = 0.5f * (cImgZB0 + cImgZB1);

	float4 accColA = make_float4(0.f, 0.f, 0.f, 0.f);
	float4 accColB = make_float4(0.f, 0.f, 0.f, 0.f);
	float4 accImgY = 0.5f * (cImgYA + cImgYB);
	for (int iy = cy - halfWinSize, winIdx = 0; iy <= cy + halfWinSize; iy++) {
		for (int ix = cx - halfWinSize; ix <= cx + halfWinSize; ix++, winIdx++) {
			int x = (ix >= xSize) ? 2 * xSize - 2 - ix : abs(ix);
			int y = (iy >= ySize) ? 2 * ySize - 2 - iy : abs(iy);
			int iIdx = y * xSize + x;

			if (ix == cx && iy == cy)
				continue;

			int iIdxA0 = bufferIdxA0 * nPix + iIdx * colorDim;
			int iIdxA1 = bufferIdxA1 * nPix + iIdx * colorDim;
			int iIdxB0 = bufferIdxB0 * nPix + iIdx * colorDim;
			int iIdxB1 = bufferIdxB1 * nPix + iIdx * colorDim;

			const float4& iImgYA0 = make_float4(subImgsY[iIdxA0 + 0], subImgsY[iIdxA0 + 1], subImgsY[iIdxA0 + 2], 0.f);
			const float4& iImgYA1 = make_float4(subImgsY[iIdxA1 + 0], subImgsY[iIdxA1 + 1], subImgsY[iIdxA1 + 2], 0.f);
			const float4& iImgYB0 = make_float4(subImgsY[iIdxB0 + 0], subImgsY[iIdxB0 + 1], subImgsY[iIdxB0 + 2], 0.f);
			const float4& iImgYB1 = make_float4(subImgsY[iIdxB1 + 0], subImgsY[iIdxB1 + 1], subImgsY[iIdxB1 + 2], 0.f);
			const float4& iImgZA0 = make_float4(subImgsZ[iIdxA0 + 0], subImgsZ[iIdxA0 + 1], subImgsZ[iIdxA0 + 2], 0.f);
			const float4& iImgZA1 = make_float4(subImgsZ[iIdxA1 + 0], subImgsZ[iIdxA1 + 1], subImgsZ[iIdxA1 + 2], 0.f);
			const float4& iImgZB0 = make_float4(subImgsZ[iIdxB0 + 0], subImgsZ[iIdxB0 + 1], subImgsZ[iIdxB0 + 2], 0.f);
			const float4& iImgZB1 = make_float4(subImgsZ[iIdxB1 + 0], subImgsZ[iIdxB1 + 1], subImgsZ[iIdxB1 + 2], 0.f);
			
			const float4 iImgYA = 0.5f * (iImgYA0 + iImgYA1);
			const float4 iImgYB = 0.5f * (iImgYB0 + iImgYB1);
			const float4 iImgZA = 0.5f * (iImgZA0 + iImgZA1);
			const float4 iImgZB = 0.5f * (iImgZB0 + iImgZB1);

			// Calculate a simple variance-based weighting (Eq. 8)
			float4 zDiffA0 = cImgZA0 - iImgZA0;
			float4 zDiffA1 = cImgZA1 - iImgZA1;
			float4 zDiffB0 = cImgZB0 - iImgZB0;
			float4 zDiffB1 = cImgZB1 - iImgZB1;
			float4 zVarA = make_float4((zDiffA0.x - zDiffA1.x) * (zDiffA0.x - zDiffA1.x), 
									   (zDiffA0.y - zDiffA1.y) * (zDiffA0.y - zDiffA1.y), 
									   (zDiffA0.z - zDiffA1.z) * (zDiffA0.z - zDiffA1.z), 0.f);
			float4 zVarB = make_float4((zDiffB0.x - zDiffB1.x) * (zDiffB0.x - zDiffB1.x), 
									   (zDiffB0.y - zDiffB1.y) * (zDiffB0.y - zDiffB1.y), 
									   (zDiffB0.z - zDiffB1.z) * (zDiffB0.z - zDiffB1.z), 0.f);
			float4 cWgtA = make_float4(expf(-paramGamma * (float)spp * zVarA.x),
									   expf(-paramGamma * (float)spp * zVarA.y),
									   expf(-paramGamma * (float)spp * zVarA.z), 0.f);
			float4 cWgtB = make_float4(expf(-paramGamma * (float)spp * zVarB.x),
									   expf(-paramGamma * (float)spp * zVarB.y),
									   expf(-paramGamma * (float)spp * zVarB.z), 0.f);
			
			accColA += cWgtA * ((cImgZA - iImgZA) - (cImgYA - iImgYA));
			accColB += cWgtB * ((cImgZB - iImgZB) - (cImgYB - iImgYB));
			accImgY += 0.5f * (iImgYA + iImgYB);
		}
	}

	float invEle = 1.f / ((float)winSizeSqr - 1.f);
	float4 outColA = make_float4(cImgYA.x + invEle * accColA.x,
								 cImgYA.y + invEle * accColA.y,
								 cImgYA.z + invEle * accColA.z, 0.f);
	float4 outColB = make_float4(cImgYB.x + invEle * accColB.x,
								 cImgYB.y + invEle * accColB.y,
								 cImgYB.z + invEle * accColB.z, 0.f);

	float4 avgImgY = (1.f / (float)winSizeSqr) * accImgY;
	float4 varNumerator = (outColA - outColB) * (outColA - outColB);
	float4 varDenominator = avgImgY * avgImgY;
	float avgDenoisedVar = (varNumerator.x / (varDenominator.x + 1e-2f)
						  + varNumerator.y / (varDenominator.y + 1e-2f)
						  + varNumerator.z / (varDenominator.z + 1e-2f)) / 3.f;

	outVar[cIdx] = avgDenoisedVar;
}

__global__ void KernelDenoising(float *outImg, float *subImgsY, float *subImgsZ,
	int xSize, int ySize, int winSize, int spp, float paramGamma) {
	const int cx = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
	const int cy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);
	if (cx >= xSize || cy >= ySize)
		return;

	const int cIdx = cy * xSize + cx;
	const int halfWinSize = winSize / 2;
	const int winSizeSqr = winSize * winSize;
	const int colorDim = 3;
	const int nPix = colorDim * xSize * ySize;

	const int bufferIdxA0 = 0;
	const int bufferIdxA1 = 2;
	const int bufferIdxB0 = 1;
	const int bufferIdxB1 = 3;

	int cIdxA0 = bufferIdxA0 * nPix + cIdx * colorDim;
	int cIdxA1 = bufferIdxA1 * nPix + cIdx * colorDim;
	int cIdxB0 = bufferIdxB0 * nPix + cIdx * colorDim;
	int cIdxB1 = bufferIdxB1 * nPix + cIdx * colorDim;

	const float4& cImgYA0 = make_float4(subImgsY[cIdxA0 + 0], subImgsY[cIdxA0 + 1], subImgsY[cIdxA0 + 2], 0.f);
	const float4& cImgYA1 = make_float4(subImgsY[cIdxA1 + 0], subImgsY[cIdxA1 + 1], subImgsY[cIdxA1 + 2], 0.f);
	const float4& cImgYB0 = make_float4(subImgsY[cIdxB0 + 0], subImgsY[cIdxB0 + 1], subImgsY[cIdxB0 + 2], 0.f);
	const float4& cImgYB1 = make_float4(subImgsY[cIdxB1 + 0], subImgsY[cIdxB1 + 1], subImgsY[cIdxB1 + 2], 0.f);
	const float4& cImgZA0 = make_float4(subImgsZ[cIdxA0 + 0], subImgsZ[cIdxA0 + 1], subImgsZ[cIdxA0 + 2], 0.f);
	const float4& cImgZA1 = make_float4(subImgsZ[cIdxA1 + 0], subImgsZ[cIdxA1 + 1], subImgsZ[cIdxA1 + 2], 0.f);
	const float4& cImgZB0 = make_float4(subImgsZ[cIdxB0 + 0], subImgsZ[cIdxB0 + 1], subImgsZ[cIdxB0 + 2], 0.f);
	const float4& cImgZB1 = make_float4(subImgsZ[cIdxB1 + 0], subImgsZ[cIdxB1 + 1], subImgsZ[cIdxB1 + 2], 0.f);
	
	const float4 cImgYA = 0.5f * (cImgYA0 + cImgYA1);
	const float4 cImgYB = 0.5f * (cImgYB0 + cImgYB1);
	const float4 cImgZA = 0.5f * (cImgZA0 + cImgZA1);
	const float4 cImgZB = 0.5f * (cImgZB0 + cImgZB1);
	const float4 cImgY = 0.5f * (cImgYA + cImgYB);
	const float4 cImgZ = 0.5f * (cImgZA + cImgZB);

	float4 accCol = make_float4(0.f, 0.f, 0.f, 0.f);
	for (int iy = cy - halfWinSize, winIdx = 0; iy <= cy + halfWinSize; iy++) {
		for (int ix = cx - halfWinSize; ix <= cx + halfWinSize; ix++, winIdx++) {
			int x = (ix >= xSize) ? 2 * xSize - 2 - ix : abs(ix);
			int y = (iy >= ySize) ? 2 * ySize - 2 - iy : abs(iy);
			int iIdx = y * xSize + x;

			if (ix == cx && iy == cy)
				continue;

			int iIdxA0 = bufferIdxA0 * nPix + iIdx * colorDim;
			int iIdxA1 = bufferIdxA1 * nPix + iIdx * colorDim;
			int iIdxB0 = bufferIdxB0 * nPix + iIdx * colorDim;
			int iIdxB1 = bufferIdxB1 * nPix + iIdx * colorDim;

			const float4& iImgYA0 = make_float4(subImgsY[iIdxA0 + 0], subImgsY[iIdxA0 + 1], subImgsY[iIdxA0 + 2], 0.f);
			const float4& iImgYA1 = make_float4(subImgsY[iIdxA1 + 0], subImgsY[iIdxA1 + 1], subImgsY[iIdxA1 + 2], 0.f);
			const float4& iImgYB0 = make_float4(subImgsY[iIdxB0 + 0], subImgsY[iIdxB0 + 1], subImgsY[iIdxB0 + 2], 0.f);
			const float4& iImgYB1 = make_float4(subImgsY[iIdxB1 + 0], subImgsY[iIdxB1 + 1], subImgsY[iIdxB1 + 2], 0.f);
			const float4& iImgZA0 = make_float4(subImgsZ[iIdxA0 + 0], subImgsZ[iIdxA0 + 1], subImgsZ[iIdxA0 + 2], 0.f);
			const float4& iImgZA1 = make_float4(subImgsZ[iIdxA1 + 0], subImgsZ[iIdxA1 + 1], subImgsZ[iIdxA1 + 2], 0.f);
			const float4& iImgZB0 = make_float4(subImgsZ[iIdxB0 + 0], subImgsZ[iIdxB0 + 1], subImgsZ[iIdxB0 + 2], 0.f);
			const float4& iImgZB1 = make_float4(subImgsZ[iIdxB1 + 0], subImgsZ[iIdxB1 + 1], subImgsZ[iIdxB1 + 2], 0.f);
			
			const float4 iImgYA = 0.5f * (iImgYA0 + iImgYA1);
			const float4 iImgYB = 0.5f * (iImgYB0 + iImgYB1);
			const float4 iImgZA = 0.5f * (iImgZA0 + iImgZA1);
			const float4 iImgZB = 0.5f * (iImgZB0 + iImgZB1);
			const float4 iImgY = 0.5f * (iImgYA + iImgYB);
			const float4 iImgZ = 0.5f * (iImgZA + iImgZB);

			// Calculate a simple variance-based weighting (Eq. 8)
			float4 zDiffA = cImgZA - iImgZA;
			float4 zDiffB = cImgZB - iImgZB;
			float4 zVar = make_float4((zDiffA.x - zDiffB.x) * (zDiffA.x - zDiffB.x),
									  (zDiffA.y - zDiffB.y) * (zDiffA.y - zDiffB.y),
									  (zDiffA.z - zDiffB.z) * (zDiffA.z - zDiffB.z), 0.f);
			float4 cWgt = make_float4(expf(-paramGamma * (float)spp * zVar.x),
									  expf(-paramGamma * (float)spp * zVar.y),
									  expf(-paramGamma * (float)spp * zVar.z), 0.f);
			
			accCol += cWgt * ((cImgZ - iImgZ) - (cImgY - iImgY));
		}
	}

	float invEle = 1.f / ((float)winSizeSqr - 1.f);
	outImg[colorDim * cIdx + 0] = cImgY.x + invEle * accCol.x;
	outImg[colorDim * cIdx + 1] = cImgY.y + invEle * accCol.y;
	outImg[colorDim * cIdx + 2] = cImgY.z + invEle * accCol.z;
}

__global__ void KernelDenoisingForL2(float *outImg, float *subImgsY, float *subImgsZ,
	int xSize, int ySize, int winSize, int spp, float paramGamma) {
	const int cx = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
	const int cy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);
	if (cx >= xSize || cy >= ySize)
		return;

	const int cIdx = cy * xSize + cx;
	const int halfWinSize = winSize / 2;
	const int winSizeSqr = winSize * winSize;
	const int colorDim = 3;
	const int nPix = colorDim * xSize * ySize;

	const int bufferIdxA0 = 0;
	const int bufferIdxA1 = 2;
	const int bufferIdxB0 = 1;
	const int bufferIdxB1 = 3;

	int cIdxA0 = bufferIdxA0 * nPix + cIdx * colorDim;
	int cIdxA1 = bufferIdxA1 * nPix + cIdx * colorDim;
	int cIdxB0 = bufferIdxB0 * nPix + cIdx * colorDim;
	int cIdxB1 = bufferIdxB1 * nPix + cIdx * colorDim;

	const float4& cImgYA0 = make_float4(subImgsY[cIdxA0 + 0], subImgsY[cIdxA0 + 1], subImgsY[cIdxA0 + 2], 0.f);
	const float4& cImgYA1 = make_float4(subImgsY[cIdxA1 + 0], subImgsY[cIdxA1 + 1], subImgsY[cIdxA1 + 2], 0.f);
	const float4& cImgYB0 = make_float4(subImgsY[cIdxB0 + 0], subImgsY[cIdxB0 + 1], subImgsY[cIdxB0 + 2], 0.f);
	const float4& cImgYB1 = make_float4(subImgsY[cIdxB1 + 0], subImgsY[cIdxB1 + 1], subImgsY[cIdxB1 + 2], 0.f);
	const float4& cImgZA0 = make_float4(subImgsZ[cIdxA0 + 0], subImgsZ[cIdxA0 + 1], subImgsZ[cIdxA0 + 2], 0.f);
	const float4& cImgZA1 = make_float4(subImgsZ[cIdxA1 + 0], subImgsZ[cIdxA1 + 1], subImgsZ[cIdxA1 + 2], 0.f);
	const float4& cImgZB0 = make_float4(subImgsZ[cIdxB0 + 0], subImgsZ[cIdxB0 + 1], subImgsZ[cIdxB0 + 2], 0.f);
	const float4& cImgZB1 = make_float4(subImgsZ[cIdxB1 + 0], subImgsZ[cIdxB1 + 1], subImgsZ[cIdxB1 + 2], 0.f);
	
	const float4 cImgYA = 0.5f * (cImgYA0 + cImgYA1);
	const float4 cImgYB = 0.5f * (cImgYB0 + cImgYB1);
	const float4 cImgZA = 0.5f * (cImgZA0 + cImgZA1);
	const float4 cImgZB = 0.5f * (cImgZB0 + cImgZB1);

	float4 accColA = make_float4(0.f, 0.f, 0.f, 0.f);
	float4 accColB = make_float4(0.f, 0.f, 0.f, 0.f);
	for (int iy = cy - halfWinSize, winIdx = 0; iy <= cy + halfWinSize; iy++) {
		for (int ix = cx - halfWinSize; ix <= cx + halfWinSize; ix++, winIdx++) {
			int x = (ix >= xSize) ? 2 * xSize - 2 - ix : abs(ix);
			int y = (iy >= ySize) ? 2 * ySize - 2 - iy : abs(iy);
			int iIdx = y * xSize + x;

			if (ix == cx && iy == cy)
				continue;

			int iIdxA0 = bufferIdxA0 * nPix + iIdx * colorDim;
			int iIdxA1 = bufferIdxA1 * nPix + iIdx * colorDim;
			int iIdxB0 = bufferIdxB0 * nPix + iIdx * colorDim;
			int iIdxB1 = bufferIdxB1 * nPix + iIdx * colorDim;

			const float4& iImgYA0 = make_float4(subImgsY[iIdxA0 + 0], subImgsY[iIdxA0 + 1], subImgsY[iIdxA0 + 2], 0.f);
			const float4& iImgYA1 = make_float4(subImgsY[iIdxA1 + 0], subImgsY[iIdxA1 + 1], subImgsY[iIdxA1 + 2], 0.f);
			const float4& iImgYB0 = make_float4(subImgsY[iIdxB0 + 0], subImgsY[iIdxB0 + 1], subImgsY[iIdxB0 + 2], 0.f);
			const float4& iImgYB1 = make_float4(subImgsY[iIdxB1 + 0], subImgsY[iIdxB1 + 1], subImgsY[iIdxB1 + 2], 0.f);
			const float4& iImgZA0 = make_float4(subImgsZ[iIdxA0 + 0], subImgsZ[iIdxA0 + 1], subImgsZ[iIdxA0 + 2], 0.f);
			const float4& iImgZA1 = make_float4(subImgsZ[iIdxA1 + 0], subImgsZ[iIdxA1 + 1], subImgsZ[iIdxA1 + 2], 0.f);
			const float4& iImgZB0 = make_float4(subImgsZ[iIdxB0 + 0], subImgsZ[iIdxB0 + 1], subImgsZ[iIdxB0 + 2], 0.f);
			const float4& iImgZB1 = make_float4(subImgsZ[iIdxB1 + 0], subImgsZ[iIdxB1 + 1], subImgsZ[iIdxB1 + 2], 0.f);
			
			const float4 iImgYA = 0.5f * (iImgYA0 + iImgYA1);
			const float4 iImgYB = 0.5f * (iImgYB0 + iImgYB1);
			const float4 iImgZA = 0.5f * (iImgZA0 + iImgZA1);
			const float4 iImgZB = 0.5f * (iImgZB0 + iImgZB1);

			// Calculate a simple variance-based weighting (Eq. 8)
			float4 zDiffA0 = cImgZA0 - iImgZA0;
			float4 zDiffA1 = cImgZA1 - iImgZA1;
			float4 zDiffB0 = cImgZB0 - iImgZB0;
			float4 zDiffB1 = cImgZB1 - iImgZB1;
			float4 zVarA = make_float4((zDiffA0.x - zDiffA1.x) * (zDiffA0.x - zDiffA1.x), 
									   (zDiffA0.y - zDiffA1.y) * (zDiffA0.y - zDiffA1.y), 
									   (zDiffA0.z - zDiffA1.z) * (zDiffA0.z - zDiffA1.z), 0.f);
			float4 zVarB = make_float4((zDiffB0.x - zDiffB1.x) * (zDiffB0.x - zDiffB1.x), 
									   (zDiffB0.y - zDiffB1.y) * (zDiffB0.y - zDiffB1.y), 
									   (zDiffB0.z - zDiffB1.z) * (zDiffB0.z - zDiffB1.z), 0.f);
			float4 cWgtA = make_float4(expf(-paramGamma  * (float)spp * zVarA.x),
									   expf(-paramGamma  * (float)spp * zVarA.y),
									   expf(-paramGamma  * (float)spp * zVarA.z), 0.f);
			float4 cWgtB = make_float4(expf(-paramGamma  * (float)spp * zVarB.x),
									   expf(-paramGamma  * (float)spp * zVarB.y),
									   expf(-paramGamma  * (float)spp * zVarB.z), 0.f);
			
			accColA += cWgtA * ((cImgZA - iImgZA) - (cImgYA - iImgYA));
			accColB += cWgtB * ((cImgZB - iImgZB) - (cImgYB - iImgYB));
		}
	}

	float invEle = 1.f / ((float)winSizeSqr - 1.f);
	float4 outColA = make_float4(cImgYA.x + invEle * accColA.x,
								 cImgYA.y + invEle * accColA.y,
								 cImgYA.z + invEle * accColA.z, 0.f);
	float4 outColB = make_float4(cImgYB.x + invEle * accColB.x,
								 cImgYB.y + invEle * accColB.y,
								 cImgYB.z + invEle * accColB.z, 0.f);

	outImg[colorDim * cIdx + 0] = 0.5f * (outColA.x + outColB.x);
	outImg[colorDim * cIdx + 1] = 0.5f * (outColA.y + outColB.y);
	outImg[colorDim * cIdx + 2] = 0.5f * (outColA.z + outColB.z);
}


float applyParameterSelection(dim3 threads, dim3 grid, float *d_outVar, float *d_subImgsY, float *d_subImgsZ,
	int xSize, int ySize, int winSize, int spp) {
	const int maxIter = 10;
	const float paramGammaSet[maxIter] = { 0.01f, 0.025f, 0.05f, 0.1f, 0.2f,
										   0.5f, 1.f, 1.5f, 2.f, 2.5f };

	int minIterIdx = -1;
	float minDenoisedVar = FLT_MAX;
	for (int iter = 0; iter < maxIter; iter++) {
		float cParamGamma = paramGammaSet[iter];

		KernelCalcDenoisedVariance << < grid, threads >> >(d_outVar, d_subImgsY, d_subImgsZ,
			xSize, ySize, winSize, spp, cParamGamma);
		hipDeviceSynchronize();

		thrust::device_ptr<float> d_outVar_ptr(d_outVar);
		float accDenoisedVar = thrust::reduce(d_outVar_ptr, d_outVar_ptr + xSize * ySize);
		float avgDenoisedVar = accDenoisedVar / (float)(xSize * ySize);

		if (avgDenoisedVar < minDenoisedVar) {
			minDenoisedVar = avgDenoisedVar;
			minIterIdx = iter;
		}
	}

	if (minIterIdx < 0)
		printf("[denoiser.cu] Please check minIterIdx!\n");

	return paramGammaSet[minIterIdx];
}

void Denoiser::allocMemory() {
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount > 1) {
		printf("[denoiser.cu] Multiple GPUs exist, which is not supported!\n");
		hipSetDevice(0);
	}

	hipMalloc((void **)&d_outImg, 3 * xSize * ySize * sizeof(float));
	hipMalloc((void **)&d_outVar, xSize * ySize * sizeof(float));

	hipMalloc((void **)&d_imgY, 3 * xSize * ySize * sizeof(float));
	hipMalloc((void **)&d_imgZ, 3 * xSize * ySize * sizeof(float));
	hipMalloc((void **)&d_subImgsY, numBuffers * 3 * xSize * ySize * sizeof(float));
	hipMalloc((void **)&d_subImgsZ, numBuffers * 3 * xSize * ySize * sizeof(float));

	h_subImgsY = new float[numBuffers * 3 * xSize * ySize];
	h_subImgsZ = new float[numBuffers * 3 * xSize * ySize];
}

void Denoiser::deallocMemory() {
	hipFree(d_outImg);
	hipFree(d_outVar);

	hipFree(d_imgY);
	hipFree(d_imgZ);
	hipFree(d_subImgsY);
	hipFree(d_subImgsZ);

	delete[] h_subImgsY;
	delete[] h_subImgsZ;
}

void Denoiser::runDenoiser(std::vector<float> &outImg,
	std::vector< std::vector<float> > &subImgsY, std::vector< std::vector<float> > &subImgsZ, int spp, bool isL2) {
	const int blockDim = 16;
	dim3 threads(blockDim, blockDim);
	dim3 grid(iDivUp(xSize, blockDim), iDivUp(ySize, blockDim));

	const int nPix = 3 * xSize * ySize;
	int nSppForCorrEst = spp / 2;
	int nSppForSubBuffer = nSppForCorrEst / numBuffers;
	
	if (isL2) {
		nSppForCorrEst = spp;
		nSppForSubBuffer = nSppForCorrEst / numBuffers;

		// JH: decorrelation
		const int bufferIdxA0 = 0, bufferIdxA1 = 2;
		const int bufferIdxB0 = 1, bufferIdxB1 = 3;
		for (int pixIdx = 0; pixIdx < nPix; pixIdx++) {
			h_subImgsY[bufferIdxA0 * nPix + pixIdx] = subImgsY[bufferIdxA0][pixIdx];
			h_subImgsY[bufferIdxA1 * nPix + pixIdx] = subImgsY[bufferIdxA1][pixIdx];
			h_subImgsY[bufferIdxB0 * nPix + pixIdx] = subImgsY[bufferIdxB0][pixIdx];
			h_subImgsY[bufferIdxB1 * nPix + pixIdx] = subImgsY[bufferIdxB1][pixIdx];

			h_subImgsZ[bufferIdxA0 * nPix + pixIdx] = subImgsZ[bufferIdxB0][pixIdx];
			h_subImgsZ[bufferIdxA1 * nPix + pixIdx] = subImgsZ[bufferIdxB1][pixIdx];
			h_subImgsZ[bufferIdxB0 * nPix + pixIdx] = subImgsZ[bufferIdxA0][pixIdx];
			h_subImgsZ[bufferIdxB1 * nPix + pixIdx] = subImgsZ[bufferIdxA1][pixIdx];
		}
	}
	else {
		for (int bufferIdx = 0; bufferIdx < numBuffers; bufferIdx++) {
			for (int pixIdx = 0; pixIdx < nPix; pixIdx++) {
				h_subImgsY[bufferIdx * nPix + pixIdx] = subImgsY[bufferIdx][pixIdx];
				h_subImgsZ[bufferIdx * nPix + pixIdx] = subImgsZ[bufferIdx][pixIdx];
			}
		}
	}

	hipMemcpy(d_subImgsY, h_subImgsY, numBuffers * nPix * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_subImgsZ, h_subImgsZ, numBuffers * nPix * sizeof(float), hipMemcpyHostToDevice);

	printf("[denoiser.cu] Denoising starts!\n");
	// ==============================================================
	// Parameter selection (last paragraph in Sec. 4.2)
	float optParamGamma = applyParameterSelection(threads, grid, d_outVar, d_subImgsY, d_subImgsZ,
		xSize, ySize, winSize, nSppForSubBuffer);
	
	// Denoising with an example kernel satisfying our conditions
	// - Simple variance-based weighting with B=2 (Eq. 8 in Sec. 4.2)
	if (isL2) {
		KernelDenoisingForL2 << <grid, threads >> >(d_outImg, d_subImgsY, d_subImgsZ,
			xSize, ySize, winSize, nSppForSubBuffer, optParamGamma);
		hipDeviceSynchronize();
	}
	else {
		int nSppForHalfBuffer = nSppForCorrEst / 2;
		KernelDenoising << <grid, threads >> >(d_outImg, d_subImgsY, d_subImgsZ,
			xSize, ySize, winSize, nSppForHalfBuffer, optParamGamma);
		hipDeviceSynchronize();
	}
	// ==============================================================
	printf("[denoiser.cu] Denoising done!\n");

	// Final output saving
	hipMemcpy(outImg.data(), d_outImg, nPix * sizeof(float), hipMemcpyDeviceToHost);
	hipGetLastError();
}